#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "histogram-equalization.h"
#include "hist-equ.h"

__global__ void rgb2hsl_convert(PPM_IMG *img_in, HSL_IMG *img_out, int *img_size) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    float L = 0.0;
    float H = 0.0;
    float S = 0.0;
	
    while(i < *img_size) {
        float var_r = ( (float)img_in->img_r[i]/255 );//Convert RGB to [0,1]
        float var_g = ( (float)img_in->img_g[i]/255 );
        float var_b = ( (float)img_in->img_b[i]/255 );
        float var_min = (var_r < var_g) ? var_r : var_g;
        var_min = (var_min < var_b) ? var_min : var_b;   //min. value of RGB
        float var_max = (var_r > var_g) ? var_r : var_g;
        var_max = (var_max > var_b) ? var_max : var_b;   //max. value of RGB
        float del_max = var_max - var_min;              //Delta RGB value
        
        L = ( var_max + var_min ) / 2;
        if ( del_max == 0 ) {                   //This is a gray, no chroma...
            H = 0;         
            S = 0;    
        } else {                                    //Chromatic data...
            if ( L < 0.5 ) {
                S = del_max/(var_max+var_min);
            } else {
                S = del_max/(2-var_max-var_min );
            }
            if(S > 1) {
                printf("S too big...\n");
            }

            float del_r = (((var_max-var_r)/6)+(del_max/2))/del_max;
            float del_g = (((var_max-var_g)/6)+(del_max/2))/del_max;
            float del_b = (((var_max-var_b)/6)+(del_max/2))/del_max;
            if( var_r == var_max ) {
                H = del_b - del_g;  
            } else {       
                if( var_g == var_max ) {
                    H = (1.0/3.0) + del_r - del_b;
                } else {
                    H = (2.0/3.0) + del_g - del_r;
                }   
            }
        }
        
        if ( H < 0 ) {
            H += 1;
        }
        if ( H > 1 ) {
            H -= 1;
        }

        __syncthreads();
        
        img_out->h[i] = H;
        img_out->s[i] = S;
        img_out->l[i] = (unsigned char)(L*255);

        i+= offset;
    }   
}

HSL_IMG rgb2hsl_gpu(PPM_IMG img_in) 
{
    int img_size = img_in.w * img_in.h;
    
    HSL_IMG img_out;
    img_out.width  = img_in.w;
    img_out.height = img_in.h;
    img_out.h = (float *)malloc(img_size * sizeof(float));
    img_out.s = (float *)malloc(img_size * sizeof(float));
    img_out.l = (unsigned char *)malloc(img_size * sizeof(unsigned char));
	

    unsigned char *l = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    float *h = (float *)malloc(img_size * sizeof(float));
    float *s = (float *)malloc(img_size * sizeof(float));

    unsigned char *img_r = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    unsigned char *img_g = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    unsigned char *img_b = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    
	PPM_IMG *img_in_d;
	HSL_IMG *img_out_d;
    int *img_size_d;
	
    hipMalloc((void**)&img_out_d, sizeof(HSL_IMG));
    hipMalloc((void**)&l, sizeof(unsigned char) * img_size);
    hipMalloc((void**)&h, sizeof(float) * img_size);
    hipMalloc((void**)&s, sizeof(float) * img_size);

    hipMalloc((void**)&img_in_d, sizeof(PPM_IMG));
    hipMalloc((void**)&img_r, sizeof(unsigned char) * img_size);
    hipMalloc((void**)&img_g, sizeof(unsigned char) * img_size);
    hipMalloc((void**)&img_b, sizeof(unsigned char) * img_size);


    hipMemcpy(img_out_d, &img_out, sizeof(HSL_IMG), hipMemcpyHostToDevice);
    hipMemcpy(l, img_out.l, sizeof(unsigned char) *img_size, hipMemcpyHostToDevice);
    hipMemcpy(h, img_out.h, sizeof(float) *img_size, hipMemcpyHostToDevice);
    hipMemcpy(s, img_out.s, sizeof(float) *img_size, hipMemcpyHostToDevice);
    hipMemcpy(&(img_out_d->l), &l, sizeof(l), hipMemcpyHostToDevice);
    hipMemcpy(&(img_out_d->h), &h, sizeof(h), hipMemcpyHostToDevice);
    hipMemcpy(&(img_out_d->s), &s, sizeof(s), hipMemcpyHostToDevice);

    hipMemcpy(img_in_d, &img_in, sizeof(PPM_IMG), hipMemcpyHostToDevice);
    hipMemcpy(img_r, img_in.img_r, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);
    hipMemcpy(img_g, img_in.img_g, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);
    hipMemcpy(img_b, img_in.img_b, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);
    hipMemcpy(&(img_in_d->img_r), &img_r, sizeof(img_r), hipMemcpyHostToDevice);
    hipMemcpy(&(img_in_d->img_g), &img_g, sizeof(img_g), hipMemcpyHostToDevice);
    hipMemcpy(&(img_in_d->img_b), &img_b, sizeof(img_b), hipMemcpyHostToDevice);


    hipMalloc((void**)&img_size_d, sizeof(int));
	hipMemcpy(img_size_d, &img_size, sizeof(int), hipMemcpyHostToDevice);
	
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int blocks = prop.multiProcessorCount;

	rgb2hsl_convert<<<blocks * 2, 1024>>>(img_in_d, img_out_d, img_size_d);
	
    hipMemcpy(img_out.l, l, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.h, h, sizeof(float) * img_size, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.s, s, sizeof(float) * img_size, hipMemcpyDeviceToHost);


	hipFree(img_in_d);
    hipFree(img_r);
    hipFree(img_g);
    hipFree(img_b);
	hipFree(img_out_d);
    hipFree(h);
    hipFree(s);
    hipFree(l);
    hipFree(img_size_d);

    return img_out;
}

__device__ float Hue_2_RGB_gpu( float v1, float v2, float vH )             //Function Hue_2_RGB
{
    if ( vH < 0 ) vH += 1;
    if ( vH > 1 ) vH -= 1;
    if ( ( 6 * vH ) < 1 ) return ( v1 + ( v2 - v1 ) * 6 * vH );
    if ( ( 2 * vH ) < 1 ) return ( v2 );
    if ( ( 3 * vH ) < 2 ) return ( v1 + ( v2 - v1 ) * ( ( 2.0f/3.0f ) - vH ) * 6 );
    return ( v1 );
}

//Convert HSL to RGB, assume H, S in [0.0, 1.0] and L in [0, 255]
//Output R,G,B in [0, 255]
__global__ void hsl2rgb_convert(HSL_IMG *img_in, PPM_IMG *img_out, int *img_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    while(i < *img_size) {
        float H = img_in->h[i];
        float S = img_in->s[i];
        float L = img_in->l[i]/255.0f;
        float var_1, var_2;
        
        unsigned char r,g,b;
        
        if ( S == 0 ) {
            r = L * 255;
            g = L * 255;
            b = L * 255;
        } else {
            if ( L < 0.5 ) {
                var_2 = L * ( 1 + S );
            } else {
                var_2 = ( L + S ) - ( S * L );
            }

            var_1 = 2 * L - var_2;
            r = 255 * Hue_2_RGB_gpu( var_1, var_2, H + (1.0f/3.0f) );
            g = 255 * Hue_2_RGB_gpu( var_1, var_2, H );
            b = 255 * Hue_2_RGB_gpu( var_1, var_2, H - (1.0f/3.0f) );
        }

        __syncthreads();

        img_out->img_r[i] = r;
        img_out->img_g[i] = g;
        img_out->img_b[i] = b;

        i += offset;
    }
}

PPM_IMG hsl2rgb_gpu(HSL_IMG img_in) 
{
    int img_size = img_in.width * img_in.height;
    
    PPM_IMG img_out;
    img_out.w = img_in.width;
    img_out.h = img_in.height;
    img_out.img_r = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    img_out.img_g = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    img_out.img_b = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    

    unsigned char *l = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    float *h = (float *)malloc(img_size * sizeof(float));
    float *s = (float *)malloc(img_size * sizeof(float));

    unsigned char *img_r = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    unsigned char *img_g = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    unsigned char *img_b = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    
    HSL_IMG *img_in_d;
    PPM_IMG *img_out_d;
    int *img_size_d;
    
    hipMalloc((void**)&img_out_d, sizeof(PPM_IMG));
    hipMalloc((void**)&img_r, sizeof(unsigned char) * img_size);
    hipMalloc((void**)&img_g, sizeof(unsigned char) * img_size);
    hipMalloc((void**)&img_b, sizeof(unsigned char) * img_size);
    

    hipMalloc((void**)&img_in_d, sizeof(HSL_IMG));
    hipMalloc((void**)&l, sizeof(unsigned char) * img_size);
    hipMalloc((void**)&h, sizeof(float) * img_size);
    hipMalloc((void**)&s, sizeof(float) * img_size);


    hipMemcpy(img_out_d, &img_out, sizeof(PPM_IMG), hipMemcpyHostToDevice);
    hipMemcpy(img_r, img_out.img_r, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);
    hipMemcpy(img_g, img_out.img_g, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);
    hipMemcpy(img_b, img_out.img_b, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);
    hipMemcpy(&(img_out_d->img_r), &img_r, sizeof(img_r), hipMemcpyHostToDevice);
    hipMemcpy(&(img_out_d->img_g), &img_g, sizeof(img_g), hipMemcpyHostToDevice);
    hipMemcpy(&(img_out_d->img_b), &img_b, sizeof(img_b), hipMemcpyHostToDevice);


    hipMemcpy(img_in_d, &img_in, sizeof(PPM_IMG), hipMemcpyHostToDevice);
    hipMemcpy(l, img_in.l, sizeof(unsigned char) *img_size, hipMemcpyHostToDevice);
    hipMemcpy(h, img_in.h, sizeof(float) *img_size, hipMemcpyHostToDevice);
    hipMemcpy(s, img_in.s, sizeof(float) *img_size, hipMemcpyHostToDevice);
    hipMemcpy(&(img_in_d->l), &l, sizeof(l), hipMemcpyHostToDevice);
    hipMemcpy(&(img_in_d->h), &h, sizeof(h), hipMemcpyHostToDevice);
    hipMemcpy(&(img_in_d->s), &s, sizeof(s), hipMemcpyHostToDevice);


    hipMalloc((void**)&img_size_d, sizeof(int));
    hipMemcpy(img_size_d, &img_size, sizeof(int), hipMemcpyHostToDevice);
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int blocks = prop.multiProcessorCount;

    hsl2rgb_convert<<<blocks * 2, 1024>>>(img_in_d, img_out_d, img_size_d);
    
    hipMemcpy(img_out.img_r, img_r, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_g, img_g, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_b, img_b, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost);


    hipFree(img_in_d);
    hipFree(img_r);
    hipFree(img_g);
    hipFree(img_b);
    hipFree(img_out_d);
    hipFree(h);
    hipFree(s);
    hipFree(l);
    hipFree(img_size_d);

    return img_out;
}